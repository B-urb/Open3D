#include "hip/hip_runtime.h"
#include <stdgpu/iterator.h>  // device_begin, device_end
#include <stdgpu/memory.h>    // createDeviceArray, destroyDeviceArray
#include <stdgpu/platform.h>  // STDGPU_HOST_DEVICE
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>

#include <iostream>
#include <stdgpu/unordered_map.cuh>  // stdgpu::unordered_map

#include "open3d/core/hashmap/Hashmap.h"
#include "open3d/utility/Console.h"
#include "open3d/utility/Timer.h"

__global__ void insert_numbers(const int* d_keys,
                               const int* d_values,
                               const stdgpu::index_t n,
                               stdgpu::unordered_map<int, int> map) {
    stdgpu::index_t i =
            static_cast<stdgpu::index_t>(blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= n) return;
    map.emplace(d_keys[i], d_values[i]);
}

__global__ void find_numbers(const int* d_keys,
                             int* d_values,
                             const stdgpu::index_t n,
                             stdgpu::unordered_map<int, int> map) {
    stdgpu::index_t i =
            static_cast<stdgpu::index_t>(blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= n) return;
    d_values[i] = map.find(d_keys[i])->second;
}

int main(int argc, char** argv) {
    //
    // EXAMPLE DESCRIPTION
    // -------------------
    // This example demonstrates how stdgpu::unordered_map is used to compute a
    // duplicate-free set of numbers.
    //
    using namespace open3d;

    stdgpu::index_t n =
            utility::GetProgramOptionAsInt(argc, argv, "--n", 10000);
    int runs = utility::GetProgramOptionAsInt(argc, argv, "--runs", 1000);

    // Ours
    core::Tensor t_keys = core::Tensor::Arange(0, n, 1, core::Dtype::Int32,
                                               core::Device("CUDA:0"));
    core::Tensor t_values = core::Tensor::Arange(0, n, 1, core::Dtype::Int32,
                                                 core::Device("CUDA:0"));

    // Warm up
    core::Device device("CUDA:0");
    {
        core::Hashmap hashmap(n, core::Dtype::Int32, core::Dtype::Int32,
                              core::SizeVector{1}, core::SizeVector{1}, device);
        core::Tensor t_addrs({n}, core::Dtype::Int32, device);
        core::Tensor t_masks({n}, core::Dtype::Bool, device);

        hashmap.Insert(t_keys, t_values, t_addrs, t_masks);
        hashmap.Find(t_keys, t_addrs, t_masks);
        hipDeviceSynchronize();
    }

    utility::Timer timer;

    double insert_time = 0;
    double find_time = 0;
    for (int i = 0; i < runs; ++i) {
        core::Hashmap hashmap(n, core::Dtype::Int32, core::Dtype::Int32,
                              core::SizeVector{1}, core::SizeVector{1}, device);
        core::Tensor t_addrs({n}, core::Dtype::Int32, device);
        core::Tensor t_masks({n}, core::Dtype::Bool, device);

        timer.Start();
        hashmap.Insert(t_keys, t_values, t_addrs, t_masks);
        hipDeviceSynchronize();
        timer.Stop();
        insert_time += timer.GetDuration();

        timer.Start();
        hashmap.Find(t_keys, t_addrs, t_masks);
        hipDeviceSynchronize();
        timer.Stop();
        find_time += timer.GetDuration();

        if (hashmap.Size() != n) {
            utility::LogError("ours: incorrect insertion");
        }
    }
    utility::LogInfo("ours takes {} on average for insertion",
                     insert_time / runs);
    utility::LogInfo("ours takes {} on average for query", find_time / runs);

    // stdgpu
    int* d_keys = createDeviceArray<int>(n);
    thrust::sequence(stdgpu::device_begin(d_keys), stdgpu::device_end(d_keys),
                     0);
    int* d_values = createDeviceArray<int>(n);
    thrust::sequence(stdgpu::device_begin(d_values),
                     stdgpu::device_end(d_values), 0);
    insert_time = 0;
    find_time = 0;
    for (int i = 0; i < runs; ++i) {
        stdgpu::unordered_map<int, int> map =
                stdgpu::unordered_map<int, int>::createDeviceObject(n);
        stdgpu::index_t threads = 128;
        stdgpu::index_t blocks = (n + threads - 1) / threads;

        timer.Start();
        insert_numbers<<<static_cast<unsigned int>(blocks),
                         static_cast<unsigned int>(threads)>>>(d_keys, d_values,
                                                               n, map);
        hipDeviceSynchronize();
        timer.Stop();
        insert_time += timer.GetDuration();

        timer.Start();
        find_numbers<<<static_cast<unsigned int>(blocks),
                       static_cast<unsigned int>(threads)>>>(d_keys, d_values,
                                                             n, map);
        hipDeviceSynchronize();
        timer.Stop();
        find_time += timer.GetDuration();

        if (map.size() != n) {
            utility::LogError("stdgpu: incorrect insertion");
        }

        stdgpu::unordered_map<int, int>::destroyDeviceObject(map);
    }
    utility::LogInfo("stdgpu takes {} on average for insertion",
                     insert_time / runs);
    utility::LogInfo("stdgpu takes {} on average for query", find_time / runs);
    destroyDeviceArray<int>(d_keys);
    destroyDeviceArray<int>(d_values);
}
