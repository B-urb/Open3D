#include "hip/hip_runtime.h"
//***************************************************************************************/
//
//    Based on Pointnet2 Library (MIT License):
//    https://github.com/sshaoshuai/Pointnet2.PyTorch
//
//    Copyright (c) 2019 Shaoshuai Shi
//
//    Permission is hereby granted, free of charge, to any person obtaining a
//    copy of this software and associated documentation files (the "Software"),
//    to deal in the Software without restriction, including without limitation
//    the rights to use, copy, modify, merge, publish, distribute, sublicense,
//    and/or sell copies of the Software, and to permit persons to whom the
//    Software is furnished to do so, subject to the following conditions:
//
//    The above copyright notice and this permission notice shall be included in
//    all copies or substantial portions of the Software.
//
//    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
//    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
//    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
//    THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
//    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
//    FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
//    DEALINGS IN THE SOFTWARE.
//
//***************************************************************************************/

#include <stdio.h>
#include <stdlib.h>

#include "ATen/cuda/HIPContext.h"
#include "open3d/ml/pytorch/pointnet/GroupPointsKernel.h"
#include "open3d/ml/pytorch/pointnet/cuda_utils.h"

__global__ void group_points_grad_kernel(int b,
                                         int c,
                                         int n,
                                         int npoints,
                                         int nsample,
                                         const float *__restrict__ grad_out,
                                         const int *__restrict__ idx,
                                         float *__restrict__ grad_points) {
    // grad_out: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      grad_points: (B, C, N)
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;
    grad_out += bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
                pt_idx * nsample + sample_idx;
    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;

    atomicAdd(grad_points + bs_idx * c * n + c_idx * n + idx[0], grad_out[0]);
}

void group_points_grad_launcher(int b,
                                int c,
                                int n,
                                int npoints,
                                int nsample,
                                const float *grad_out,
                                const int *idx,
                                float *grad_points) {
    // grad_out: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      grad_points: (B, C, N)
    hipError_t err;

    auto stream = at::cuda::getCurrentCUDAStream();

    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
                b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel<<<blocks, threads, 0, stream>>>(
            b, c, n, npoints, nsample, grad_out, idx, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

__global__ void group_points_kernel(int b,
                                    int c,
                                    int n,
                                    int npoints,
                                    int nsample,
                                    const float *__restrict__ points,
                                    const int *__restrict__ idx,
                                    float *__restrict__ out) {
    // points: (B, C, N)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints, nsample)
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;

    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    int in_idx = bs_idx * c * n + c_idx * n + idx[0];
    int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
                  pt_idx * nsample + sample_idx;

    out[out_idx] = points[in_idx];
}

void group_points_launcher(int b,
                           int c,
                           int n,
                           int npoints,
                           int nsample,
                           const float *points,
                           const int *idx,
                           float *out) {
    // points: (B, C, N)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints, nsample)
    hipError_t err;

    auto stream = at::cuda::getCurrentCUDAStream();

    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
                b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel<<<blocks, threads, 0, stream>>>(
            b, c, n, npoints, nsample, points, idx, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
